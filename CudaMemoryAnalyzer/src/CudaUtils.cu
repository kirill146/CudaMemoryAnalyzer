#include <iostream>
#include "AnalyzerException.h"
#include <hip/hip_runtime.h>

size_t BufSizeByAddress(void const* p) {
	hipDeviceptr_t pbase;
	size_t psize;
	hipError_t res = hipMemGetAddressRange(&pbase, &psize, (hipDeviceptr_t)p);
	if (res != hipSuccess) {
		std::cout << "hipMemGetAddressRange() failed\n";
		throw AnalyzerException("Invalid pointer to buffer in kernel call");
	}
	uint64_t offset = (uint64_t)p - pbase;
	return psize - offset;
}
