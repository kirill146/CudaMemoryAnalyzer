#include "CudaBuffer.h"
#include <hip/hip_runtime.h>
#include "AnalyzerException.h"

CudaBuffer::CudaBuffer(size_t size)
    : buf(nullptr)
{
    hipError_t err = hipMalloc(&buf, size);
    if (err != hipSuccess) {
        throw AnalyzerException("Cannot allocate device memory");
    }
}

void* CudaBuffer::Get() const {
    return buf;
}

CudaBuffer::~CudaBuffer() {
    hipFree(buf);
}
